#include "hip/hip_runtime.h"

 /* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// *********************************************************************
// A simple demo application that implements a
// vector dot product computation between 2 float arrays. 
//
// Runs computations with on the GPU device and then checks results 
// against basic host CPU/C++ computation.
// *********************************************************************

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "shrUtils.h"
#include <omp.h>

// Forward Declarations
void DotProductHost(const float* pfData1, const float* pfData2, float* pfResult, int iNumElements);

__global__
void dot_product(const float *__restrict__ a,
                 const float *__restrict__ b,
                       float *__restrict__ c,
#ifdef ASYNC
                 const int streamIdx,
                 const int pivot,
#endif
                 const int n,
                 const int iKWeight)
{
  int iGID = blockIdx.x * blockDim.x + threadIdx.x;

#ifdef ASYNC
  iGID += pivot +streamIdx*n;
  if (iGID < pivot +(streamIdx+1)*n) {
#else
  if (iGID < n) {
#endif

    int iInOffset = iGID << 2;
    for (int k = 0; k < iKWeight; k++) 
    c[iGID] = a[iInOffset    ] * b[iInOffset    ] +
              a[iInOffset + 1] * b[iInOffset + 1] +
              a[iInOffset + 2] * b[iInOffset + 2] +
              a[iInOffset + 3] * b[iInOffset + 3];
  }
}

int main(int argc, char **argv)
{
#ifdef ASYNC
  if (argc >= 9) {
#else
  if (argc != 5) {
#endif
    printf("Usage: %s <number of elements> <repeat>\n", argv[0]);
    return 1;
  }
  const int iNumElements = atoi(argv[1]);
  const int iNumIterations = atoi(argv[2]);
  const int iKWeight = atoi(argv[3]);
  // set and log Global and Local work size dimensions
  int szLocalWorkSize = atoi(argv[4]);
#ifdef ASYNC
  const int ncustreams = atoi(argv[5]);
  const int nhostthreads  = atoi(argv[6]);
  const int fraction  = atoi(argv[7]); //partition for the computation cpu-gpu
  const int numElementsCPU = iNumElements/fraction;
  const int numElementsGPU = iNumElements - numElementsCPU;
  const int numElements_stream = numElementsGPU/ncustreams;
  const size_t src_size_cpu = numElementsCPU *4;
  const size_t dst_gpu_size_bytes = numElementsGPU * sizeof(float);
  const size_t src_size_stream = numElements_stream *4;
  const size_t src_size_bytes_stream  = src_size_stream* sizeof(float);
#endif
  // rounded up to the nearest multiple of the LocalWorkSize
  int szGlobalWorkSize = shrRoundUp((int)szLocalWorkSize, iNumElements);  

  const size_t src_size = szGlobalWorkSize * 4;
  const size_t src_size_bytes = src_size * sizeof(float);

  const size_t dst_size = szGlobalWorkSize;
  const size_t dst_size_bytes = dst_size * sizeof(float);

  // Allocate and initialize host arrays
float* srcA;
float* srcB;

#ifdef ASYNC
  hipHostMalloc (&srcA, src_size_bytes);
  hipHostMalloc (&srcB, src_size_bytes);
#else
  srcA = (float*) malloc (src_size_bytes);
  srcB = (float*) malloc (src_size_bytes);
#endif
  float*  dst = (float*) malloc (dst_size_bytes);

  float* Golden = (float*) malloc (sizeof(float) * iNumElements);
  shrFillArray(srcA, 4 * iNumElements);
  shrFillArray(srcB, 4 * iNumElements);

  float *d_srcA;
  float *d_srcB;
  float *d_dst; 

  hipMalloc((void**)&d_srcA, src_size_bytes);
  hipMalloc((void**)&d_srcB, src_size_bytes);
  hipMalloc((void**)&d_dst, dst_size_bytes);

  //printf("Global Work Size \t\t= %d\nLocal Work Size \t\t= %d\n# of Work Groups \t\t= %d\n\n", 
      //szGlobalWorkSize, szLocalWorkSize, (szGlobalWorkSize % szLocalWorkSize + szGlobalWorkSize/szLocalWorkSize)); 
#ifdef ASYNC
  dim3 grid (numElements_stream % szLocalWorkSize + numElements_stream/szLocalWorkSize); 
#else
  dim3 grid (szGlobalWorkSize % szLocalWorkSize + szGlobalWorkSize/szLocalWorkSize); 
#endif
  dim3 block (szLocalWorkSize);

  //hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

#ifdef ASYNC
  hipStream_t custream[ncustreams];
  for (int ics=0; ics<ncustreams; ics++ )
    hipStreamCreate(&custream[ics]);
#endif

  for (int i = 0; i < iNumIterations; i++) {
#ifdef ASYNC
    //hipMemcpyAsync(d_srcA, srcA, src_size_bytes, hipMemcpyHostToDevice, custream[0]);
    //hipMemcpyAsync(d_srcB, srcB, src_size_bytes, hipMemcpyHostToDevice, custream[1]);
    //hipDeviceSynchronize();

    DotProductHost ((const float*)srcA, (const float*)srcB, (float*)dst, numElementsCPU);
    #pragma omp parallel num_threads( nhostthreads)
    {
       //hipStream_t custream;
       //hipStreamCreate(&custream);
       //int tid = omp_get_thread_num();
       //int offset = tid*ncustreams_thread;
    #pragma omp for nowait
    for (int k=0; k<ncustreams; k++){
   //      hipStream_t custream;
   //      hipStreamCreate(&custream);
         size_t offset = src_size_cpu +k*src_size_stream;
         hipMemcpyAsync(&d_srcA[offset], &srcA[offset], src_size_bytes_stream, hipMemcpyHostToDevice, custream[k]);
         hipMemcpyAsync(&d_srcB[offset], &srcB[offset], src_size_bytes_stream, hipMemcpyHostToDevice, custream[k]);
         hipDeviceSynchronize();

       dot_product<<<grid, block, 0, custream[k] >>>(d_srcA, d_srcB, d_dst, k, numElementsCPU, numElements_stream, iKWeight);

      // hipStreamDestroy(custream);
    }
    }

#else
    hipMemcpy(d_srcA, srcA, src_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_srcB, srcB, src_size_bytes, hipMemcpyHostToDevice);
    dot_product<<<grid, block>>>(d_srcA, d_srcB, d_dst, iNumElements, iKWeight);
#endif

}
  hipDeviceSynchronize();
#ifdef ASYNC
  hipMemcpy(&dst[numElementsCPU], &d_dst[numElementsCPU], dst_gpu_size_bytes, hipMemcpyDeviceToHost);
  for (int ics=0; ics<ncustreams; ics++ )
    hipStreamDestroy(custream[ics]);
#else
  hipMemcpy(dst, d_dst, dst_size_bytes, hipMemcpyDeviceToHost);
#endif

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  //printf("Average execution time %f (s)\n", (time * 1e-9f) / iNumIterations);
  printf("%f\n", (time * 1e-9f) );

  // Compute and compare results for golden-host and report errors and pass/fail
  //printf("Comparing against Host/C++ computation...\n\n"); 
  DotProductHost ((const float*)srcA, (const float*)srcB, (float*)Golden, iNumElements);
  shrBOOL bMatch = shrComparefet((const float*)Golden, (const float*)dst, (unsigned int)iNumElements, 0.0f, 0);
  //printf("\nGPU Result %s CPU Result\n", (bMatch == shrTRUE) ? "matches" : "DOESN'T match"); 

#ifdef ASYNC
  hipHostFree(srcA);
  hipHostFree(srcB);
#endif

  hipFree(d_dst);
  hipFree(d_srcA);
  hipFree(d_srcB);

#ifndef ASYNC
  free(srcA);
  free(srcB);
#endif

  free(dst);
  free(Golden);
  return EXIT_SUCCESS;
}

// "Golden" Host processing dot product function for comparison purposes
// *********************************************************************
void DotProductHost(const float* pfData1, const float* pfData2, float* pfResult, int iNumElements)
{
  int i, j, k;
#pragma omp parallel for
  for (i = 0, j = 0; i < iNumElements; i++) 
  {
    pfResult[i] = 0.0f;
    for (k = 0; k < 4; k++, j++) 
    {
      pfResult[i] += pfData1[j] * pfData2[j]; 
    } 
  }
}
