#include "hip/hip_runtime.h"

 /* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// *********************************************************************
// A simple demo application that implements a
// vector dot product computation between 2 int arrays. 
//
// Runs computations with on the GPU device and then checks results 
// against basic host CPU/C++ computation.
// *********************************************************************

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "shrUtils.h"
#include <omp.h>

#define etype float

// Forward Declarations
void DotProductHost(const etype* pfData1, const etype* pfData2, etype* pfResult, int iNumElements, int iKWeight);

__global__
void dot_product(const etype *__restrict__ a,
                 const etype *__restrict__ b,
                       etype *__restrict__ c,
#ifdef ASYNC
                 const int streamIdx,
#endif
                 const int pivot,
                 const int n,
                 const int iKWeight)
{
  int iGID = blockIdx.x * blockDim.x + threadIdx.x;

  iGID += pivot;
#ifdef ASYNC
  iGID += streamIdx*n;
  if (iGID < pivot +(streamIdx+1)*n) {
#else
  if (iGID < n) {
#endif

    int iInOffset = iGID << 2;
    for (int k = 0; k < iKWeight; k++) 
    c[iGID] = a[iInOffset    ] * b[iInOffset    ] +
              a[iInOffset + 1] * b[iInOffset + 1] +
              a[iInOffset + 2] * b[iInOffset + 2] +
              a[iInOffset + 3] * b[iInOffset + 3];
  }
}

int main(int argc, char **argv)
{
#ifdef ASYNC
  if (argc >= 9) {
#else
  if (argc != 6) {
#endif
    printf("Usage: %s <number of elements> <repeat>\n", argv[0]);
    return 1;
  }
  const int iNumElements = atoi(argv[1]);
  const int iNumIterations = atoi(argv[2]);
  const int iKWeight = atoi(argv[3]);
  // set and log Global and Local work size dimensions
  int szLocalWorkSize = atoi(argv[4]);
  const int fraction  = atoi(argv[5]); //partition for the computation cpu-gpu
  const int numElementsCPU = iNumElements/fraction;
  const int numElementsGPU = iNumElements - numElementsCPU;
  const size_t src_size_cpu = numElementsCPU *4;
  const size_t dst_gpu_size_bytes = numElementsGPU * sizeof(etype);
  const size_t src_gpu_size_bytes = numElementsGPU *4* sizeof(etype);
#ifdef ASYNC
  const int ncustreams = atoi(argv[6]);
  const int nhostthreads  = atoi(argv[7]);
  const int numElements_stream = numElementsGPU/ncustreams;
  const size_t src_size_stream = numElements_stream *4;
  const size_t src_size_bytes_stream  = src_size_stream* sizeof(etype);
#endif
  // rounded up to the nearest multiple of the LocalWorkSize
  int szGlobalWorkSize = shrRoundUp((int)szLocalWorkSize, iNumElements);  

  const size_t src_size = szGlobalWorkSize * 4;
  const size_t src_size_bytes = src_size * sizeof(etype);

  const size_t dst_size = szGlobalWorkSize;
  const size_t dst_size_bytes = dst_size * sizeof(etype);

  // Allocate and initialize host arrays
etype* srcA;
etype* srcB;

#ifdef ASYNC
  hipHostMalloc (&srcA, src_size_bytes);
  hipHostMalloc (&srcB, src_size_bytes);
#else
  srcA = (etype*) malloc (src_size_bytes);
  srcB = (etype*) malloc (src_size_bytes);
#endif
  etype*  dst = (etype*) malloc (dst_size_bytes);

  etype* Golden = (etype*) malloc (sizeof(etype) * iNumElements);
  shrFillArray(srcA, 4 * iNumElements);
  shrFillArray(srcB, 4 * iNumElements);

  etype *d_srcA;
  etype *d_srcB;
  etype *d_dst; 

  hipMalloc((void**)&d_srcA, src_size_bytes);
  hipMalloc((void**)&d_srcB, src_size_bytes);
  hipMalloc((void**)&d_dst, dst_size_bytes);

  //printf("Global Work Size \t\t= %d\nLocal Work Size \t\t= %d\n# of Work Groups \t\t= %d\n\n", 
      //szGlobalWorkSize, szLocalWorkSize, (szGlobalWorkSize % szLocalWorkSize + szGlobalWorkSize/szLocalWorkSize)); 
#ifdef ASYNC
  dim3 grid (numElements_stream % szLocalWorkSize + numElements_stream/szLocalWorkSize); 
#else
  dim3 grid (numElementsGPU % szLocalWorkSize + numElementsGPU/szLocalWorkSize); 
#endif
  dim3 block (szLocalWorkSize);

  //hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

#ifdef ASYNC
  hipStream_t custream[ncustreams];
  for (int ics=0; ics<ncustreams; ics++ )
    hipStreamCreate(&custream[ics]);
#endif

  for (int i = 0; i < iNumIterations; i++) {
#ifdef ASYNC
    //hipMemcpyAsync(d_srcA, srcA, src_size_bytes, hipMemcpyHostToDevice, custream[0]);
    //hipMemcpyAsync(d_srcB, srcB, src_size_bytes, hipMemcpyHostToDevice, custream[1]);
    //hipDeviceSynchronize();

    DotProductHost ((const etype*)srcA, (const etype*)srcB, (etype*)dst, numElementsCPU, iKWeight);
    #pragma omp parallel num_threads( nhostthreads)
    {
       //hipStream_t custream;
       //hipStreamCreate(&custream);
       //int tid = omp_get_thread_num();
       //int offset = tid*ncustreams_thread;
    #pragma omp for nowait
    for (int k=0; k<ncustreams; k++){
   //      hipStream_t custream;
   //      hipStreamCreate(&custream);
//int k = 0;
         size_t offset = src_size_cpu +k*src_size_stream;
         hipMemcpyAsync(&d_srcA[offset], &srcA[offset], src_size_bytes_stream, hipMemcpyHostToDevice, custream[k]);
         hipMemcpyAsync(&d_srcB[offset], &srcB[offset], src_size_bytes_stream, hipMemcpyHostToDevice, custream[k]);
  //       hipDeviceSynchronize();
    //}
//}
       dot_product<<<grid, block, 0, custream[k] >>>(d_srcA, d_srcB, d_dst, k, numElementsCPU, numElements_stream, iKWeight);

      // hipStreamDestroy(custream);
    }
    }

#else
    size_t offset = src_size_cpu;
    hipMemcpy(&d_srcA[offset], &srcA[offset], src_gpu_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(&d_srcB[offset], &srcB[offset], src_gpu_size_bytes, hipMemcpyHostToDevice);
    dot_product<<<grid, block>>>(d_srcA, d_srcB, d_dst, numElementsCPU, iNumElements, iKWeight);
    DotProductHost ((const etype*)srcA, (const etype*)srcB, (etype*)dst, numElementsCPU, iKWeight);
#endif

}
  hipDeviceSynchronize();
#ifdef ASYNC
  hipMemcpy(&dst[numElementsCPU], &d_dst[numElementsCPU], dst_gpu_size_bytes, hipMemcpyDeviceToHost);
  for (int ics=0; ics<ncustreams; ics++ )
    hipStreamDestroy(custream[ics]);
#else
  hipMemcpy(&dst[numElementsCPU], &d_dst[numElementsCPU], dst_gpu_size_bytes, hipMemcpyDeviceToHost);
  //hipMemcpy(dst, d_dst, dst_size_bytes, hipMemcpyDeviceToHost);
#endif

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  //printf("Average execution time %f (s)\n", (time * 1e-9f) / iNumIterations);
  printf("%f\n", (time * 1e-9f) );

  // Compute and compare results for golden-host and report errors and pass/fail
  //printf("Comparing against Host/C++ computation...\n\n"); 
  DotProductHost ((const etype*)srcA, (const etype*)srcB, (etype*)Golden, iNumElements, iKWeight);
  shrBOOL bMatch = shrComparefet((const etype*)Golden, (const etype*)dst, (unsigned int)iNumElements, 0.0f, 0);
  printf("\nGPU Result %s CPU Result\n", (bMatch == shrTRUE) ? "matches" : "DOESN'T match"); 

#ifdef ASYNC
  hipHostFree(srcA);
  hipHostFree(srcB);
#endif

  hipFree(d_dst);
  hipFree(d_srcA);
  hipFree(d_srcB);

#ifndef ASYNC
  free(srcA);
  free(srcB);
#endif

  free(dst);
  free(Golden);
  return EXIT_SUCCESS;
}

// "Golden" Host processing dot product function for comparison purposes
// *********************************************************************
void DotProductHost(const etype* pfData1, const etype* pfData2, etype* pfResult, int iNumElements, int iKWeight)
{
  int i, j, k;
#pragma omp parallel for 
  for (i = 0; i < iNumElements; i++) 
  {
    j = 4*i;
    for (int wl = 0; wl < iKWeight; wl++) {
      pfResult[i] = pfData1[j] * pfData2[j]
                  + pfData1[j+1] * pfData2[j+1] 
                  + pfData1[j+2] * pfData2[j+2]
                  + pfData1[j+3] * pfData2[j+3]; 
  }
  }
}
