#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

//define the data set size (cubic volume)
#define DATAXSIZE 600
#define DATAYSIZE 600
#define DATAZSIZE 600

// square
#define SQ(x) ((x)*(x))

__device__
double dFphi(double phi, double u, double lambda)
{
  return (-phi*(1.0-phi*phi)+lambda*u*(1.0-phi*phi)*(1.0-phi*phi));
}

__device__
double GradientX(double phi[][DATAYSIZE][DATAXSIZE], 
                 double dx, double dy, double dz, int x, int y, int z)
{
  return (phi[x+1][y][z] - phi[x-1][y][z]) / (2.0*dx);
}

__device__
double GradientY(double phi[][DATAYSIZE][DATAXSIZE], 
                 double dx, double dy, double dz, int x, int y, int z)
{
  return (phi[x][y+1][z] - phi[x][y-1][z]) / (2.0*dy);
}

__device__
double GradientZ(double phi[][DATAYSIZE][DATAXSIZE], 
                 double dx, double dy, double dz, int x, int y, int z)
{
  return (phi[x][y][z+1] - phi[x][y][z-1]) / (2.0*dz);
}

__device__
double Divergence(double phix[][DATAYSIZE][DATAXSIZE], 
                  double phiy[][DATAYSIZE][DATAXSIZE],
                  double phiz[][DATAYSIZE][DATAXSIZE], 
                  double dx, double dy, double dz, int x, int y, int z)
{
  return GradientX(phix,dx,dy,dz,x,y,z) + 
         GradientY(phiy,dx,dy,dz,x,y,z) +
         GradientZ(phiz,dx,dy,dz,x,y,z);
}

__device__
double Laplacian(double phi[][DATAYSIZE][DATAXSIZE],
                 double dx, double dy, double dz, int x, int y, int z)
{
  double phixx = (phi[x+1][y][z] + phi[x-1][y][z] - 2.0 * phi[x][y][z]) / SQ(dx);
  double phiyy = (phi[x][y+1][z] + phi[x][y-1][z] - 2.0 * phi[x][y][z]) / SQ(dy);
  double phizz = (phi[x][y][z+1] + phi[x][y][z-1] - 2.0 * phi[x][y][z]) / SQ(dz);
  return phixx + phiyy + phizz;
}

__device__
double An(double phix, double phiy, double phiz, double epsilon)
{
  if (phix != 0.0 || phiy != 0.0 || phiz != 0.0){
    return ((1.0 - 3.0 * epsilon) * (1.0 + (((4.0 * epsilon) / (1.0-3.0*epsilon))*
           ((SQ(phix)*SQ(phix)+SQ(phiy)*SQ(phiy)+SQ(phiz)*SQ(phiz)) /
           ((SQ(phix)+SQ(phiy)+SQ(phiz))*(SQ(phix)+SQ(phiy)+SQ(phiz)))))));
  }
  else
  {
    return (1.0-((5.0/3.0)*epsilon));
  }
}

__device__
double Wn(double phix, double phiy, double phiz, double epsilon, double W0)
{
  return (W0*An(phix,phiy,phiz,epsilon));
}

__device__
double taun(double phix, double phiy, double phiz, double epsilon, double tau0)
{
  return tau0 * SQ(An(phix,phiy,phiz,epsilon));
}

__device__
double dFunc(double l, double m, double n)
{
  if (l != 0.0 || m != 0.0 || n != 0.0){
    return (((l*l*l*(SQ(m)+SQ(n)))-(l*(SQ(m)*SQ(m)+SQ(n)*SQ(n)))) /
            ((SQ(l)+SQ(m)+SQ(n))*(SQ(l)+SQ(m)+SQ(n))));
  }
  else
  {
    return 0.0;
  }
}

__global__
void calculateForce(double phi[][DATAYSIZE][DATAXSIZE], 
                    double Fx[][DATAYSIZE][DATAXSIZE],
                    double Fy[][DATAYSIZE][DATAXSIZE],
                    double Fz[][DATAYSIZE][DATAXSIZE],
                    double dx, double dy, double dz,
                    double epsilon, double W0, double tau0)
{

  unsigned iz = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned ix = blockIdx.z*blockDim.z + threadIdx.z;

  if ((ix < (DATAXSIZE-1)) && (iy < (DATAYSIZE-1)) && 
      (iz < (DATAZSIZE-1)) && (ix > (0)) && 
      (iy > (0)) && (iz > (0))) {

    double phix = GradientX(phi,dx,dy,dz,ix,iy,iz);
    double phiy = GradientY(phi,dx,dy,dz,ix,iy,iz);
    double phiz = GradientZ(phi,dx,dy,dz,ix,iy,iz);
    double sqGphi = SQ(phix) + SQ(phiy) + SQ(phiz);
    double c = 16.0 * W0 * epsilon;
    double w = Wn(phix,phiy,phiz,epsilon,W0);
    double w2 = SQ(w);
    

    Fx[ix][iy][iz] = w2 * phix + sqGphi * w * c * dFunc(phix,phiy,phiz);
    Fy[ix][iy][iz] = w2 * phiy + sqGphi * w * c * dFunc(phiy,phiz,phix);
    Fz[ix][iy][iz] = w2 * phiz + sqGphi * w * c * dFunc(phiz,phix,phiy);
  }
  else
  {
    Fx[ix][iy][iz] = 0.0;
    Fy[ix][iy][iz] = 0.0;
    Fz[ix][iy][iz] = 0.0;
  }

}

// device function to set the 3D volume
__global__
void allenCahn(double phinew[][DATAYSIZE][DATAXSIZE], 
               double phiold[][DATAYSIZE][DATAXSIZE],
               double uold[][DATAYSIZE][DATAXSIZE],
               double Fx[][DATAYSIZE][DATAXSIZE],
               double Fy[][DATAYSIZE][DATAXSIZE],
               double Fz[][DATAYSIZE][DATAXSIZE],
               double epsilon, double W0, double tau0, double lambda,
               double dt, double dx, double dy, double dz)
{
  unsigned iz = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned ix = blockIdx.z*blockDim.z + threadIdx.z;

  if ((ix < (DATAXSIZE-1)) && (iy < (DATAYSIZE-1)) && 
      (iz < (DATAZSIZE-1)) && (ix > (0)) && 
      (iy > (0)) && (iz > (0))) {

    double phix = GradientX(phiold,dx,dy,dz,ix,iy,iz);
    double phiy = GradientY(phiold,dx,dy,dz,ix,iy,iz);
    double phiz = GradientZ(phiold,dx,dy,dz,ix,iy,iz); 

    phinew[ix][iy][iz] = phiold[ix][iy][iz] + 
     (dt / taun(phix,phiy,phiz,epsilon,tau0)) * 
     (Divergence(Fx,Fy,Fz,dx,dy,dz,ix,iy,iz) - 
      dFphi(phiold[ix][iy][iz], uold[ix][iy][iz],lambda));
  }
}

__global__
void boundaryConditionsPhi(double phinew[][DATAYSIZE][DATAXSIZE])
{
  unsigned iz = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned ix = blockIdx.z*blockDim.z + threadIdx.z;

  if (ix == 0){
    phinew[ix][iy][iz] = -1.0;
  }
  else if (ix == DATAXSIZE-1){
    phinew[ix][iy][iz] = -1.0;
  }
  else if (iy == 0){
    phinew[ix][iy][iz] = -1.0;
  }
  else if (iy == DATAYSIZE-1){
    phinew[ix][iy][iz] = -1.0;
  }
  else if (iz == 0){
    phinew[ix][iy][iz] = -1.0;
  }
  else if (iz == DATAZSIZE-1){
    phinew[ix][iy][iz] = -1.0;
  }
}

__global__
void thermalEquation(double unew[][DATAYSIZE][DATAXSIZE],
                     double uold[][DATAYSIZE][DATAXSIZE],
                     double phinew[][DATAYSIZE][DATAXSIZE],
                     double phiold[][DATAYSIZE][DATAXSIZE],
                     double D, double dt, double dx, double dy, double dz)
{
  unsigned iz = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned ix = blockIdx.z*blockDim.z + threadIdx.z;

  if ((ix < (DATAXSIZE-1)) && (iy < (DATAYSIZE-1)) && 
      (iz < (DATAZSIZE-1)) && (ix > (0)) && 
      (iy > (0)) && (iz > (0))){
    unew[ix][iy][iz] = uold[ix][iy][iz] + 
      0.5*(phinew[ix][iy][iz]- phiold[ix][iy][iz]) +
      dt * D * Laplacian(uold,dx,dy,dz,ix,iy,iz);
  }
}

__global__
void boundaryConditionsU(double unew[][DATAYSIZE][DATAXSIZE], double delta)
{
  unsigned iz = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned ix = blockIdx.z*blockDim.z + threadIdx.z;

  if (ix == 0){
    unew[ix][iy][iz] =  -delta;
  }
  else if (ix == DATAXSIZE-1){
    unew[ix][iy][iz] =  -delta;
  }
  else if (iy == 0){
    unew[ix][iy][iz] =  -delta;
  }
  else if (iy == DATAYSIZE-1){
    unew[ix][iy][iz] =  -delta;
  }
  else if (iz == 0){
    unew[ix][iy][iz] =  -delta;
  }
  else if (iz == DATAZSIZE-1){
    unew[ix][iy][iz] =  -delta;
  }
}

__global__
void swapGrid(double cnew[][DATAYSIZE][DATAXSIZE],
              double cold[][DATAYSIZE][DATAXSIZE])
{
  unsigned iz = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned iy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned ix = blockIdx.z*blockDim.z + threadIdx.z;

  if ((ix < (DATAXSIZE)) && 
      (iy < (DATAYSIZE)) &&
      (iz < (DATAZSIZE))) {
    double tmp = cnew[ix][iy][iz];
    cnew[ix][iy][iz] = cold[ix][iy][iz];
    cold[ix][iy][iz] = tmp;
  }
}

void initializationPhi(double phi[][DATAYSIZE][DATAXSIZE], double r0)
{
  #pragma omp parallel for collapse(3)
  for (int idx = 0; idx < DATAXSIZE; idx++) {
    for (int idy = 0; idy < DATAYSIZE; idy++) {
      for (int idz = 0; idz < DATAZSIZE; idz++) {
        double r = std::sqrt(SQ(idx-0.5*DATAXSIZE) + SQ(idy-0.5*DATAYSIZE) + SQ(idz-0.5*DATAZSIZE));
        if (r < r0){
          phi[idx][idy][idz] = 1.0;
        }
        else
        {
          phi[idx][idy][idz] = -1.0;
        }
      }
    }
  }
}

void initializationU(double u[][DATAYSIZE][DATAXSIZE], double r0, double delta)
{
  #pragma omp parallel for collapse(3)
  for (int idx = 0; idx < DATAXSIZE; idx++) {
    for (int idy = 0; idy < DATAYSIZE; idy++) {
      for (int idz = 0; idz < DATAZSIZE; idz++) {
        double r = std::sqrt(SQ(idx-0.5*DATAXSIZE) + SQ(idy-0.5*DATAYSIZE) + SQ(idz-0.5*DATAZSIZE));
        if (r < r0) {
          u[idx][idy][idz] = 0.0;
        }
        else
        {
          u[idx][idy][idz] = -delta * (1.0 - std::exp(-(r-r0)));
        }
      }
    }
  }
}

int main(int argc, char *argv[])
{
  typedef double nRarray[DATAYSIZE][DATAXSIZE];
  const int num_steps = atoi(argv[1]);  //6000;
  const double dx = 0.4;
  const double dy = 0.4;
  const double dz = 0.4;
  const double dt = 0.01;
  const double delta = 0.8;
  const double r0 = 5.0;
  const double epsilon = 0.07;
  const double W0 = 1.0;
  const double beta0 = 0.0;
  const double D = 2.0;
  const double d0 = 0.5;
  const double a1 = 1.25 / std::sqrt(2.0);
  const double a2 = 0.64;
  const double lambda = (W0*a1)/(d0);
  const double tau0 = ((W0*W0*W0*a1*a2)/(d0*D)) + ((W0*W0*beta0)/(d0));

  // overall data set sizes
  const int nx = DATAXSIZE;
  const int ny = DATAYSIZE;
  const int nz = DATAZSIZE;
  // pointers for data set storage via malloc
  nRarray *phi_host; // storage for result stored on host
  nRarray *u_host;
  nRarray *d_phiold;  // storage for result computed on device
  nRarray *d_phinew;
  nRarray *d_uold;
  nRarray *d_unew;
  nRarray *d_Fx;
  nRarray *d_Fy;
  nRarray *d_Fz;
  // allocate storage for data set

  phi_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double));
  u_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double));

  // compute result

  initializationPhi(phi_host,r0);
  initializationU(u_host,r0,delta);

  double clock_d = double(clock()) / CLOCKS_PER_SEC;

  // define the chunk sizes that each threadblock will work on
  dim3 grid ((DATAZSIZE+7)/8, (DATAYSIZE+7)/8, (DATAXSIZE+3)/4);
  dim3 block (8, 8, 4);

  // allocate GPU device buffers
  hipMalloc((void **) &d_phiold, (nx*ny*nz)*sizeof(double));
  hipMalloc((void **) &d_phinew, (nx*ny*nz)*sizeof(double));
  hipMalloc((void **) &d_uold, (nx*ny*nz)*sizeof(double));
  hipMalloc((void **) &d_unew, (nx*ny*nz)*sizeof(double));
  hipMalloc((void **) &d_Fx, (nx*ny*nz)*sizeof(double));
  hipMalloc((void **) &d_Fy, (nx*ny*nz)*sizeof(double));
  hipMalloc((void **) &d_Fz, (nx*ny*nz)*sizeof(double));

  hipMemcpy(d_phiold, phi_host, ((nx*ny*nz)*sizeof(double)), hipMemcpyHostToDevice);
  hipMemcpy(d_uold, u_host, ((nx*ny*nz)*sizeof(double)), hipMemcpyHostToDevice);

  int t = 0;

  while (t <= num_steps) {

    calculateForce<<<grid, block>>>(d_phiold,d_Fx,d_Fy,d_Fz,
                                    dx,dy,dz,epsilon,W0,tau0);

    allenCahn<<<grid, block>>>(d_phinew,d_phiold,d_uold,
                               d_Fx,d_Fy,d_Fz,
                               epsilon,W0,tau0,lambda,
                               dt,dx,dy,dz);

    boundaryConditionsPhi<<<grid, block>>>(d_phinew);

    thermalEquation<<<grid, block>>>(d_unew,d_uold,d_phinew,d_phiold,
                                     D,dt,dx,dy,dz);

    boundaryConditionsU<<<grid, block>>>(d_unew,delta);

    swapGrid<<<grid, block>>>(d_phinew, d_phiold);

    swapGrid<<<grid, block>>>(d_unew, d_uold);

    t++;
  }

  hipMemcpy(phi_host, d_phiold, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
  hipMemcpy(u_host, d_uold, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);

  hipFree(d_phiold);
  hipFree(d_phinew);
  hipFree(d_uold);
  hipFree(d_unew);
  hipFree(d_Fx);
  hipFree(d_Fy);
  hipFree(d_Fz);

  clock_d = double(clock()) / CLOCKS_PER_SEC - clock_d; 
  printf("Offload time = %.3fms\n", clock_d*1e3);

#ifdef DEBUG
  for (int idx = 0; idx < nx; idx++)
    for (int idy = 0; idy < ny; idy++)
      for (int idz = 0; idz < nz; idz++)
        printf("%lf %lf\n", phi_host[idx][idy][idz], u_host[idx][idy][idz]);
#endif

  free(phi_host);
  free(u_host);

  return 0;
}
